#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hipblas.h>
#include <hipsolver.h>

#include "helper.h"
#include "dr-bcg.h"

void print_matrix(const float *mat, const int rows, const int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%6.3f ", mat[i * cols + j]);
        }
        std::cout << std::endl;
    }
}

/*
 * function [X_final, iterations] = DR_BCG(A, B, X, tol, maxit)
 *     iterations = 0;
 *     R = B - A * X;
 *     [w, sigma] = qr(R,'econ');
 *     s = w;

 *     for k = 1:maxit
 *         iterations = iterations + 1;
 *         xi = (s' * A * s)^-1;
 *         X = X + s * xi * sigma;
 *         if (norm(B(:,1) - A * X(:,1)) / norm(B(:,1))) < tol
 *             break
 *         else
 *             [w, zeta] = qr(w - A * s * xi,'econ');
 *             s = w + s * zeta';
 *             sigma = zeta * sigma;
 *         end
 *     end
 *     X_final = X;
 * end
 */
int dr_bcg(
    float *A,
    const int n,
    const float *x,
    const float *b,
    const float tolerance,
    const int max_iterations)
{
    int iterations = 0;

    hipblasHandle_t cublasH;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    // r = b - Ax as GEMM:
    // r = -1.0 * Ax + r where r initially contains b
    const float alpha = -1;
    const float beta = 1;

    float *h_r = (float *)malloc(n * sizeof(float));
    float *d_A = nullptr;
    float *d_x = nullptr;
    float *d_r = nullptr;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * n * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(float) * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_r), sizeof(float) * n));

    CUDA_CHECK(hipMemcpy(d_A, A, sizeof(float) * n * n, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_r, b, sizeof(float) * n, hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasSgemv(
        cublasH,
        HIPBLAS_OP_N,
        n,
        n,
        &alpha,
        d_A, n,
        d_x, 1,
        &beta,
        d_r, 1));

    CUDA_CHECK(hipMemcpy(h_r, d_r, sizeof(float) * n, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_r));

    std::cout << "\nAfter r = b - Ax\n"
              << std::endl;
    std::cout << "A:" << std::endl;
    print_matrix(A, n, n);
    std::cout << "x:" << std::endl;
    print_matrix(x, n, 1);
    std::cout << "r:" << std::endl;
    print_matrix(h_r, n, 1);

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    free(h_r);

    std::cout << "[INFO]Starting QR procedure [w, sigma] = qr(r)" << std::endl;
    std::vector<float> w(n * n);
    std::vector<float> sigma(n * n);
    qr_decomposition(w.data(), sigma.data(), n, A, b);

    std::cout << "\nAfter [w, sigma] = qr(r)\n"
              << std::endl;
    std::cout << "w:" << std::endl;
    print_matrix(w.data(), n, n);
    std::cout << "sigma:" << std::endl;
    print_matrix(sigma.data(), n, n);

    return iterations;
}

void qr_decomposition(float *q, float *r, const int n, float *A, const float *b)
{
    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnParams_t params = NULL;

    using data_type = float;

    std::vector<data_type> tau(n, 0);
    int info = 0;

    data_type *d_A = nullptr;
    data_type *d_b = nullptr;
    data_type *d_tau = nullptr;
    int *d_info = nullptr;

    size_t lwork_geqrf_d = 0;
    void *d_work = nullptr;
    size_t lwork_geqrf_h = 0;
    void *h_work = nullptr;

    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * n * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b), sizeof(data_type) * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_tau), sizeof(data_type) * tau.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_A, A, sizeof(data_type) * n * n, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, sizeof(data_type) * n, hipMemcpyHostToDevice));

    CUSOLVER_CHECK(hipsolverDnXgeqrf_bufferSize(cusolverH, params, n, n, HIP_R_32F, d_A,
                                               n, HIP_R_32F, d_tau,
                                               HIP_R_32F, &lwork_geqrf_d,
                                               &lwork_geqrf_h));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), lwork_geqrf_d));

    if (0 < lwork_geqrf_h)
    {
        h_work = reinterpret_cast<void *>(malloc(lwork_geqrf_h));
        if (h_work == nullptr)
        {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    CUSOLVER_CHECK(hipsolverDnXgeqrf(cusolverH, params, n, n, HIP_R_32F, d_A,
                                    n, HIP_R_32F, d_tau,
                                    HIP_R_32F, d_work, lwork_geqrf_d, h_work,
                                    lwork_geqrf_h, d_info));

    // Copy R (stored in upper triangular)
    CUDA_CHECK(hipMemcpy(r, d_A, sizeof(data_type) * n * n, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemcpy(tau.data(), d_tau, sizeof(data_type) * tau.size(), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));

    if (0 > info)
    {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    CUDA_CHECK(hipMemcpy(A, d_A, sizeof(data_type) * n * n, hipMemcpyDeviceToHost));

    // Explicitly compute Q
    int lwork_orgqr = 0;
    CUSOLVER_CHECK(hipsolverDnSorgqr_bufferSize(cusolverH, n, n, n, d_A, n, d_tau, &lwork_orgqr));
    CUSOLVER_CHECK(hipsolverDnSorgqr(cusolverH, n, n, n, d_A, n, d_tau, reinterpret_cast<float *>(d_work), lwork_orgqr, d_info));

    // Copy Q
    CUDA_CHECK(hipMemcpy(q, d_A, sizeof(data_type) * n * n, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_tau));
    CUDA_CHECK(hipFree(d_work));

    free(h_work);

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
}

void fill_random(float *mat, const int rows, const int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            mat[i * cols + j] = rand() % 100 / 100.0;
        }
    }
}

int main(int argc, char *argv[])
{
    constexpr int n = 16;
    constexpr float tolerance = 0.001;
    constexpr int max_iterations = 100;

    float *A = (float *)malloc(n * n * sizeof(float));
    fill_random(A, n, n);
    float *x = (float *)malloc(n * sizeof(float));
    float *b = (float *)malloc(n * sizeof(float));
    fill_random(b, n, 1);

    dr_bcg(A, n, x, b, tolerance, max_iterations);

    free(A);
    free(x);
    free(b);

    return 0;
}