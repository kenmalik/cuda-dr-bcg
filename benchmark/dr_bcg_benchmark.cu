#include "hip/hip_runtime.h"
#include <functional>
#include <iostream>
#include <algorithm>

#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <nvtx3/nvtx3.hpp>

#include <benchmark/benchmark.h>

#include <string>

#include "dr_bcg/dr_bcg.h"
#include "dr_bcg/helper.h"
#include "dr_bcg/device_buffer.h"

#define TIME_CUDA(function)                     \
    do                                          \
    {                                           \
        hipEvent_t start, stop;                \
        hipEventCreate(&start);                \
        hipEventCreate(&stop);                 \
                                                \
        hipEventRecord(start);                 \
        function;                               \
        hipEventRecord(stop);                  \
        hipEventSynchronize(stop);             \
                                                \
        float ms = 0;                           \
        hipEventElapsedTime(&ms, start, stop); \
        state.SetIterationTime(ms / 1000.0);    \
    } while (0);

class BenchmarkData
{
public:
    static BenchmarkData &get_data()
    {
        static BenchmarkData data;
        return data;
    }

    static int get_m()
    {
        return BenchmarkData::get_data().m;
    }

    static const float *get_A()
    {
        return BenchmarkData::get_data().d_A;
    }

    void load_bin(std::string matrix_bin_file)
    {
        std::vector<double> buffer = read_matrix_bin(matrix_bin_file);
        std::vector<float> h_A(buffer.size());
        std::transform(buffer.begin(), buffer.end(), h_A.begin(), [](double d)
                       { return static_cast<float>(d); });

        set_d_A(h_A);
        m = std::sqrt(h_A.size()); // Assume square matrix
    }

    void load_random(const int n)
    {
        std::vector<float> h_A(n * n);
        fill_spd(h_A.data(), n);

        set_d_A(h_A);
        m = n;
    }

private:
    float *d_A;
    int m;

    BenchmarkData()
    {
    }

    ~BenchmarkData()
    {
        if (d_A)
        {
            hipFree(d_A);
        }
    }

    void set_d_A(const std::vector<float> &h_A)
    {
        if (d_A)
        {
            hipFree(d_A);
        }
        CUDA_CHECK(hipMalloc(&d_A, h_A.size() * sizeof(float)));
        CUDA_CHECK(hipMemcpy(d_A, h_A.data(), h_A.size() * sizeof(float), hipMemcpyHostToDevice));
    }
};

static const std::vector<int64_t> INPUT_DIMENSION_RANGE = benchmark::CreateRange(2048, 8192, 2);
static const std::vector<int64_t> BLOCK_SIZE_RANGE = []()
{
    auto list = benchmark::CreateDenseRange(2, 16, 2);
    list.insert(list.begin(), 1);
    return list;
}();

class DR_BCG_Benchmark : public benchmark::Fixture
{
protected:
    hipblasHandle_t cublasH = NULL;
    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnParams_t cusolverParams = NULL;

    DR_BCG_Benchmark()
    {
        CUBLAS_CHECK(hipblasCreate(&cublasH));
        CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
        CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverParams));
    }

    ~DR_BCG_Benchmark()
    {
        hipblasDestroy(cublasH);
        hipsolverDnDestroy(cusolverH);
        hipsolverDnDestroyParams(cusolverParams);
    }

    std::tuple<float *, float *> initialize_inputs(const int m, const int n)
    {
        float *d_X = nullptr;
        float *d_B = nullptr;

        std::vector<float> X(m * n, 0);
        std::vector<float> B(m * n);
        fill_random(B.data(), m, n);

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_X), sizeof(float) * m * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * m * n));

        CUDA_CHECK(hipMemcpy(d_X, X.data(), sizeof(float) * m * n, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B, B.data(), sizeof(float) * m * n, hipMemcpyHostToDevice));

        return {d_X, d_B};
    }

    DeviceBuffer filled_device_buffer(
        hipsolverHandle_t &cusolverH, hipsolverDnParams_t &cusolverParams, hipblasHandle_t &cublasH,
        const int m, const int n,
        const float *d_A, float *d_X, float *d_B)
    {
        DeviceBuffer d(m, n);

        float *d_R;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(float) * m * n));

        // R = B - AX
        dr_bcg::get_R(cublasH, d_R, m, n, d_A, d_X, d_B);
        dr_bcg::qr_factorization(cusolverH, cusolverParams, d.w, d.sigma, m, n, d_R);
        CUDA_CHECK(hipFree(d_R)); // Never used later

        // s = w
        CUDA_CHECK(hipMemcpy(d.s, d.w, sizeof(float) * m * n, hipMemcpyDeviceToDevice));

        return d;
    }
};

BENCHMARK_DEFINE_F(DR_BCG_Benchmark, DR_BCG)(benchmark::State &state)
{
    constexpr float tolerance = 1e-6;
    constexpr int max_iterations = 2048;

    int iterations = 0;

    const float *d_A = BenchmarkData::get_A();
    const int m = BenchmarkData::get_m();
    const int n = state.range(0);
    auto [d_X, d_B] = initialize_inputs(m, n);

    CUDA_CHECK(hipDeviceSynchronize());

    for (auto _ : state)
    {
        TIME_CUDA(dr_bcg::dr_bcg(cusolverH, cusolverParams, cublasH, m, n, d_A, d_X, d_B, tolerance, max_iterations, &iterations));
    }

    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_B));

    state.counters["performed_algorithm_iterations"] = iterations;
    state.counters["max_algorithm_iterations"] = max_iterations;
}
BENCHMARK_REGISTER_F(DR_BCG_Benchmark, DR_BCG)
    ->MinWarmUpTime(1.0)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond)
    ->ArgsProduct({BLOCK_SIZE_RANGE});

BENCHMARK_DEFINE_F(DR_BCG_Benchmark, get_xi)(benchmark::State &state)
{
    const int m = BenchmarkData::get_m();
    const int n = state.range(0);

    const float *d_A = BenchmarkData::get_A();
    auto [d_X, d_B] = initialize_inputs(m, n);
    nvtx3::mark("get_xi (" + std::to_string(m) + ", " + std::to_string(n) + ")");

    DeviceBuffer d = filled_device_buffer(cusolverH, cusolverParams, cublasH, m, n, d_A, d_X, d_B);
    CUDA_CHECK(hipDeviceSynchronize());

    for (auto _ : state)
    {
        TIME_CUDA(dr_bcg::get_xi(cusolverH, cusolverParams, cublasH, m, n, d, d_A));
    }

    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_B));
}
BENCHMARK_REGISTER_F(DR_BCG_Benchmark, get_xi)
    ->MinWarmUpTime(1.0)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond)
    ->ArgsProduct({BLOCK_SIZE_RANGE});

BENCHMARK_DEFINE_F(DR_BCG_Benchmark, get_next_X)(benchmark::State &state)
{
    const float *d_A = BenchmarkData::get_A();
    const int m = BenchmarkData::get_m();
    const int n = state.range(0);
    auto [d_X, d_B] = initialize_inputs(m, n);
    nvtx3::mark("get_next_X (" + std::to_string(m) + ", " + std::to_string(n) + ")");

    DeviceBuffer d = filled_device_buffer(cusolverH, cusolverParams, cublasH, m, n, d_A, d_X, d_B);
    CUDA_CHECK(hipDeviceSynchronize());

    dr_bcg::get_xi(cusolverH, cusolverParams, cublasH, m, n, d, d_A);

    // Keep copy of X for consistent benchmark state
    std::vector<float> h_X(m * n);
    CUDA_CHECK(hipMemcpy(h_X.data(), d_X, sizeof(float) * h_X.size(), hipMemcpyDeviceToHost));

    for (auto _ : state)
    {
        CUDA_CHECK(hipMemcpy(d_X, h_X.data(), sizeof(float) * h_X.size(), hipMemcpyHostToDevice));
        TIME_CUDA(dr_bcg::get_next_X(cublasH, m, n, d.s, d.xi, d.temp, d.sigma, d_X));
    }

    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_B));
}
BENCHMARK_REGISTER_F(DR_BCG_Benchmark, get_next_X)
    ->MinWarmUpTime(1.0)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond)
    ->ArgsProduct({BLOCK_SIZE_RANGE});

BENCHMARK_DEFINE_F(DR_BCG_Benchmark, get_w_zeta)(benchmark::State &state)
{
    const float *d_A = BenchmarkData::get_A();
    const int m = BenchmarkData::get_m();
    const int n = state.range(0);
    auto [d_X, d_B] = initialize_inputs(m, n);
    nvtx3::mark("get_w_zeta (" + std::to_string(m) + ", " + std::to_string(n) + ")");

    DeviceBuffer d = filled_device_buffer(cusolverH, cusolverParams, cublasH, m, n, d_A, d_X, d_B);
    CUDA_CHECK(hipDeviceSynchronize());

    dr_bcg::get_xi(cusolverH, cusolverParams, cublasH, m, n, d, d_A);
    dr_bcg::get_next_X(cublasH, m, n, d.s, d.xi, d.temp, d.sigma, d_X);

    // Keep copies of w and zeta for consistent benchmark state
    std::vector<float> h_w(m * n);
    std::vector<float> h_zeta(n * n);
    CUDA_CHECK(hipMemcpy(h_w.data(), d.w, sizeof(float) * h_w.size(), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_zeta.data(), d.zeta, sizeof(float) * h_zeta.size(), hipMemcpyDeviceToHost));

    for (auto _ : state)
    {
        CUDA_CHECK(hipMemcpy(d.w, h_w.data(), sizeof(float) * h_w.size(), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d.zeta, h_zeta.data(), sizeof(float) * h_zeta.size(), hipMemcpyHostToDevice));
        TIME_CUDA(dr_bcg::get_w_zeta(cusolverH, cusolverParams, cublasH, m, n, d, d_A));
    }

    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_B));
}
BENCHMARK_REGISTER_F(DR_BCG_Benchmark, get_w_zeta)
    ->MinWarmUpTime(1.0)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond)
    ->ArgsProduct({BLOCK_SIZE_RANGE});

BENCHMARK_DEFINE_F(DR_BCG_Benchmark, get_s)(benchmark::State &state)
{
    const float *d_A = BenchmarkData::get_A();
    const int m = BenchmarkData::get_m();
    const int n = state.range(0);
    auto [d_X, d_B] = initialize_inputs(m, n);
    nvtx3::mark("get_s (" + std::to_string(m) + ", " + std::to_string(n) + ")");

    DeviceBuffer d = filled_device_buffer(cusolverH, cusolverParams, cublasH, m, n, d_A, d_X, d_B);
    CUDA_CHECK(hipDeviceSynchronize());

    dr_bcg::get_xi(cusolverH, cusolverParams, cublasH, m, n, d, d_A);
    dr_bcg::get_next_X(cublasH, m, n, d.s, d.xi, d.temp, d.sigma, d_X);
    dr_bcg::get_w_zeta(cusolverH, cusolverParams, cublasH, m, n, d, d_A);

    // Keep copy of s for consistent benchmark state
    std::vector<float> h_s(m * n);
    CUDA_CHECK(hipMemcpy(h_s.data(), d.s, sizeof(float) * h_s.size(), hipMemcpyDeviceToHost));

    for (auto _ : state)
    {
        CUDA_CHECK(hipMemcpy(d.s, h_s.data(), sizeof(float) * h_s.size(), hipMemcpyHostToDevice));
        TIME_CUDA(dr_bcg::get_s(cublasH, m, n, d));
    }

    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_B));
}
BENCHMARK_REGISTER_F(DR_BCG_Benchmark, get_s)
    ->MinWarmUpTime(1.0)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond)
    ->ArgsProduct({BLOCK_SIZE_RANGE});

BENCHMARK_DEFINE_F(DR_BCG_Benchmark, get_sigma)(benchmark::State &state)
{
    const float *d_A = BenchmarkData::get_A();
    const int m = BenchmarkData::get_m();
    const int n = state.range(0);
    auto [d_X, d_B] = initialize_inputs(m, n);
    nvtx3::mark("get_sigma (" + std::to_string(m) + ", " + std::to_string(n) + ")");

    DeviceBuffer d = filled_device_buffer(cusolverH, cusolverParams, cublasH, m, n, d_A, d_X, d_B);
    CUDA_CHECK(hipDeviceSynchronize());

    dr_bcg::get_xi(cusolverH, cusolverParams, cublasH, m, n, d, d_A);
    dr_bcg::get_next_X(cublasH, m, n, d.s, d.xi, d.temp, d.sigma, d_X);
    dr_bcg::get_w_zeta(cusolverH, cusolverParams, cublasH, m, n, d, d_A);
    dr_bcg::get_s(cublasH, m, n, d);

    // Keep copies of zeta and sigma for consistent benchmark state
    std::vector<float> h_zeta(n * n);
    std::vector<float> h_sigma(n * n);
    CUDA_CHECK(hipMemcpy(h_zeta.data(), d.zeta, sizeof(float) * h_zeta.size(), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_sigma.data(), d.sigma, sizeof(float) * h_sigma.size(), hipMemcpyDeviceToHost));

    for (auto _ : state)
    {
        CUDA_CHECK(hipMemcpy(d.zeta, h_zeta.data(), sizeof(float) * h_zeta.size(), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d.sigma, h_sigma.data(), sizeof(float) * h_sigma.size(), hipMemcpyHostToDevice));
        TIME_CUDA(dr_bcg::get_sigma(cublasH, n, d));
    }

    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_B));
}
BENCHMARK_REGISTER_F(DR_BCG_Benchmark, get_sigma)
    ->MinWarmUpTime(1.0)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond)
    ->ArgsProduct({BLOCK_SIZE_RANGE});

class QR_Benchmark : public benchmark::Fixture
{
};

BENCHMARK_DEFINE_F(QR_Benchmark, qr_factorization)(benchmark::State &state)
{
    const int m = state.range(0);
    const int n = state.range(1);

    std::vector<float> h_A(m * n);
    fill_random(h_A.data(), m, n);

    float *d_A = nullptr;
    CUDA_CHECK(hipMalloc(&d_A, sizeof(float) * h_A.size()));
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), sizeof(float) * h_A.size(), hipMemcpyHostToDevice));

    float *d_Q = nullptr;
    CUDA_CHECK(hipMalloc(&d_Q, sizeof(float) * m * n));

    float *d_R = nullptr;
    CUDA_CHECK(hipMalloc(&d_R, sizeof(float) * n * n));

    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnParams_t cusolverParams = NULL;

    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverParams));

    for (auto _ : state)
    {
        TIME_CUDA(dr_bcg::qr_factorization(cusolverH, cusolverParams, d_Q, d_R, m, n, d_A));
    }
}
BENCHMARK_REGISTER_F(QR_Benchmark, qr_factorization)
    ->MinWarmUpTime(1.0)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond)
    ->ArgsProduct({INPUT_DIMENSION_RANGE,
                   BLOCK_SIZE_RANGE});

BENCHMARK_DEFINE_F(QR_Benchmark, thin_qr)(benchmark::State &state)
{
    const int m = state.range(0);
    const int n = state.range(1);

    std::vector<float> h_A(m * n);
    fill_random(h_A.data(), m, n);

    float *d_A = nullptr;
    CUDA_CHECK(hipMalloc(&d_A, sizeof(float) * h_A.size()));
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), sizeof(float) * h_A.size(), hipMemcpyHostToDevice));

    float *d_Q = nullptr;
    CUDA_CHECK(hipMalloc(&d_Q, sizeof(float) * m * n));

    float *d_R = nullptr;
    CUDA_CHECK(hipMalloc(&d_R, sizeof(float) * n * n));

    hipblasHandle_t cublasH = NULL;
    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnParams_t cusolverParams = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverParams));

    for (auto _ : state)
    {
        TIME_CUDA(dr_bcg::thin_qr(cusolverH, cusolverParams, cublasH, d_Q, d_R, m, n, d_A));
    }
}
BENCHMARK_REGISTER_F(QR_Benchmark, thin_qr)
    ->MinWarmUpTime(1.0)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond)
    ->ArgsProduct({INPUT_DIMENSION_RANGE,
                   BLOCK_SIZE_RANGE});

void add_context()
{
    int device = 0;
    CUDA_CHECK(hipGetDevice(&device));

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));

    benchmark::AddCustomContext("device", prop.name);
    benchmark::AddCustomContext("compute_capability", std::to_string(prop.major) + "." + std::to_string(prop.minor));
}

int main(int argc, char **argv)
{
    add_context();

    benchmark::MaybeReenterWithoutASLR(argc, argv);

    std::vector<char *> args;
    std::string data_file;
    for (int i = 0; i < argc; i++)
    {
        std::string arg = argv[i];
        args.push_back(argv[i]);
        if (arg == "-d")
        {
            if (i + 1 < argc)
            {
                args.pop_back();
                data_file = argv[i + 1];
                for (int j = i; j < argc - 2; j++)
                {
                    argv[j] = argv[j + 2];
                }
                argc -= 2;
            }
            else
            {
                std::cerr << "Data file requires a value" << std::endl;
                return 1;
            }
        }
    }

    benchmark::Initialize(&argc, argv);
    if (benchmark::ReportUnrecognizedArguments(argc, argv))
    {
        return 1;
    };

    if (!data_file.empty())
    {
        BenchmarkData::get_data().load_bin(data_file);
    }
    else
    {
        std::cerr << "Data file not specified, using randomly generated matrix" << std::endl;
        constexpr int n = 4096;
        BenchmarkData::get_data().load_random(n);
    }

    benchmark::RunSpecifiedBenchmarks();
    benchmark::Shutdown();
}
