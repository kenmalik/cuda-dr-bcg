#include "hip/hip_runtime.h"
#include <tuple>
#include <iostream>
#include <vector>
#include <string>
#include <cmath>
#include <limits>

#include <suitesparse_matrix.h>

#include "dr_bcg/dr_bcg.h"
#include "dr_bcg/helper.h"

__global__ void set_val(float *A_d, float val, size_t num_elements)
{
    const int idx = blockIdx.x * blockDim.y + threadIdx.x;
    if (idx < num_elements)
    {
        A_d[idx] = val;
    }
}

int main(int argc, char *argv[])
{
    int s;
    try
    {
        if (argc == 2)
        {
            s = 1;
        }
        else if (argc == 3)
        {
            s = std::atoi(argv[2]);
        }
        else
        {
            throw std::invalid_argument("Invalid arg count");
        }
    }
    catch (const std::exception &e)
    {
        std::cerr << "Usage: ./example_2 [.mat file] [block size]" << std::endl;
        return 1;
    }

    hipsolverHandle_t cusolverH;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    hipsolverDnParams_t cusolverP;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverP));

    hipblasHandle_t cublasH;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    hipsparseHandle_t cusparseH = NULL;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseH));

    const std::string matrix_file = argv[1];
    SuiteSparseMatrix ssm(matrix_file, {"Problem"}, "A");

    int64_t *jc_d = nullptr;
    int64_t *ir_d = nullptr;
    float *vals_d = nullptr;

    float *x_d = nullptr;
    CUDA_CHECK(hipMalloc(&x_d, sizeof(float) * ssm.rows()));

    float *b_d = nullptr;
    std::vector<float> b_h(ssm.rows(), 1);
    CUDA_CHECK(hipMalloc(&b_d, sizeof(float) * b_h.size()));
    CUDA_CHECK(hipMemcpy(b_d, b_h.data(), sizeof(float) * b_h.size(), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&jc_d, sizeof(int64_t) * ssm.jc_size()));
    CUDA_CHECK(hipMalloc(&ir_d, sizeof(int64_t) * ssm.ir_size()));
    CUDA_CHECK(hipMalloc(&vals_d, sizeof(float) * ssm.nnz()));

    // Convert from default Matlab types
    std::vector<int64_t> jc_64i(ssm.jc_size());
    for (int i = 0; i < ssm.jc_size(); i++)
    {
        jc_64i[i] = static_cast<int64_t>(ssm.jc()[i]);
    }
    CUDA_CHECK(hipMemcpy(jc_d, jc_64i.data(), sizeof(int64_t) * jc_64i.size(), hipMemcpyHostToDevice));

    std::vector<int64_t> ir_64i(ssm.ir_size());
    for (int i = 0; i < ssm.ir_size(); i++)
    {
        ir_64i[i] = static_cast<int64_t>(ssm.ir()[i]);
    }
    CUDA_CHECK(hipMemcpy(ir_d, ir_64i.data(), sizeof(int64_t) * ir_64i.size(), hipMemcpyHostToDevice));

    std::vector<float> nonzeros_32f(ssm.nnz());
    for (int i = 0; i < ssm.nnz(); i++)
    {
        nonzeros_32f[i] = static_cast<float>(ssm.data()[i]);
    }
    CUDA_CHECK(hipMemcpy(vals_d, nonzeros_32f.data(), sizeof(float) * nonzeros_32f.size(), hipMemcpyHostToDevice));

    hipsparseSpMatDescr_t A;
    CUSPARSE_CHECK(hipsparseCreateCsr(
        &A, ssm.rows(), ssm.cols(), ssm.nnz(),
        jc_d, ir_d, vals_d, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    const int n = ssm.rows();

    hipsparseDnMatDescr_t X;
    float *d_X = nullptr;
    CUDA_CHECK(hipMalloc(&d_X, sizeof(float) * n * s));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&X, n, s, n, d_X, HIP_R_32F, HIPSPARSE_ORDER_COL));

    hipsparseDnMatDescr_t B;
    float *d_B = nullptr;
    CUDA_CHECK(hipMalloc(&d_B, sizeof(float) * n * s));

    constexpr int block_size = 256;
    const size_t num_elements = n * s;
    const size_t num_blocks = (num_elements + block_size - 1) / block_size;
    set_val<<<num_blocks, block_size>>>(d_B, 1, num_elements);

    CUSPARSE_CHECK(hipsparseCreateDnMat(&B, n, s, n, d_B, HIP_R_32F, HIPSPARSE_ORDER_COL));

    constexpr float tolerance = std::numeric_limits<float>::epsilon();
    constexpr int max_iterations = 10000;

    std::cout << "n: " << n << std::endl;
    std::cout << "s: " << s << std::endl;

    std::cerr << "Running..." << std::endl;
    int iterations = 0;
    dr_bcg::dr_bcg(cusolverH, cusolverP, cublasH, cusparseH, A, X, B, tolerance, max_iterations, &iterations);
    std::cerr << "Finished!" << std::endl;

    // Verification
    hipsparseDnMatDescr_t B_check;
    float *B_check_d = nullptr;
    CUDA_CHECK(hipMalloc(&B_check_d, sizeof(float) * n * s));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&B_check, n, s, n, B_check_d, HIP_R_32F, HIPSPARSE_ORDER_COL));

    constexpr hipsparseOperation_t transpose = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    constexpr float alpha = 1;
    constexpr float beta = 0;

    void *buffer = nullptr;
    size_t buffer_size = 0;

    CUSPARSE_CHECK(hipsparseSpMM_bufferSize(
        cusparseH, transpose, transpose,
        &alpha, A, X, &beta, B_check,
        HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &buffer_size));

    if (buffer_size > 0)
    {
        CUDA_CHECK(hipMalloc(&buffer, buffer_size));
    }

    CUSPARSE_CHECK(hipsparseSpMM(
        cusparseH, transpose, transpose,
        &alpha, A, X, &beta, B_check,
        HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, buffer));

    if (buffer)
    {
        CUDA_CHECK(hipFree(buffer));
    }

    std::vector<float> B_expected(n * s, 1);
    std::vector<float> B_got(n * s);
    CUDA_CHECK(hipMemcpy(B_got.data(), B_check_d, sizeof(float) * B_got.size(), hipMemcpyDeviceToHost));

    constexpr float check_tolerance = 0.001;
    float min_error = std::numeric_limits<float>::max();
    float max_error = 0;
    float avg_error = 0;

    int bad_count = 0;
    int good_count = 0;
    for (int i = 0; i < B_expected.size(); ++i)
    {
        const float error = std::abs(B_expected.at(i) - B_got.at(i));
        if (error < min_error)
        {
            min_error = error;
        }
        if (error > max_error)
        {
            max_error = error;
        }
        avg_error += error;

        if (error > check_tolerance)
        {
            ++bad_count;
        }
        else
        {
            ++good_count;
        }
    }

    std::cout << "Iterations: " << iterations << std::endl;

    std::cout << "\nWith check_tolerance=" << check_tolerance << ':' << std::endl;
    std::cout << "  Good values: " << good_count << std::endl;
    std::cout << "  Bad values: " << bad_count << std::endl;

    std::cout << "\nSummary:" << std::endl;
    std::cout << "  min_error=" << min_error << std::endl;
    std::cout << "  max_error=" << max_error << std::endl;
    std::cout << "  avg_error=" << avg_error / B_expected.size() << std::endl;

    CUDA_CHECK(hipFree(B_check_d));
    CUDA_CHECK(hipFree(jc_d));
    CUDA_CHECK(hipFree(ir_d));
    CUDA_CHECK(hipFree(vals_d));
    CUDA_CHECK(hipFree(x_d));
    CUDA_CHECK(hipFree(b_d));

    CUSPARSE_CHECK(hipsparseDestroySpMat(A));
    CUSPARSE_CHECK(hipsparseDestroyDnMat(X));
    CUSPARSE_CHECK(hipsparseDestroyDnMat(B));
    CUSPARSE_CHECK(hipsparseDestroyDnMat(B_check));

    CUSPARSE_CHECK(hipsparseDestroy(cusparseH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUSOLVER_CHECK(hipsolverDnDestroyParams(cusolverP));

    return 0;
}
