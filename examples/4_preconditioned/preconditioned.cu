#include "hip/hip_runtime.h"
#include <tuple>
#include <iostream>
#include <vector>
#include <string>
#include <cmath>
#include <limits>

#include <suitesparse_matrix.h>

#include "dr_bcg/dr_bcg.h"
#include "dr_bcg/helper.h"

__global__ void set_val(float *A_d, float val, size_t num_elements)
{
    const int idx = blockIdx.x * blockDim.y + threadIdx.x;
    if (idx < num_elements)
    {
        A_d[idx] = val;
    }
}

int main(int argc, char *argv[])
{
    int s;
    try
    {
        if (argc == 3)
        {
            s = 1;
        }
        else if (argc == 4)
        {
            s = std::atoi(argv[2]);
        }
        else
        {
            throw std::invalid_argument("Invalid arg count");
        }
    }
    catch (const std::exception &e)
    {
        std::cerr << "Usage: ./example_2 [spd matrix] [preconditioner] [block size]" << std::endl;
        return 1;
    }

    hipsolverHandle_t cusolverH;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    hipsolverDnParams_t cusolverP;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverP));

    hipblasHandle_t cublasH;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    hipsparseHandle_t cusparseH = NULL;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseH));

    const std::string matrix_file = argv[1];
    SuiteSparseMatrix ssm(matrix_file);

    int64_t *jc_d = nullptr;
    int64_t *ir_d = nullptr;
    float *vals_d = nullptr;

    float *x_d = nullptr;
    CUDA_CHECK(hipMalloc(&x_d, sizeof(float) * ssm.rows()));

    float *b_d = nullptr;
    std::vector<float> b_h(ssm.rows(), 1);
    CUDA_CHECK(hipMalloc(&b_d, sizeof(float) * b_h.size()));
    CUDA_CHECK(hipMemcpy(b_d, b_h.data(), sizeof(float) * b_h.size(), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&jc_d, sizeof(int64_t) * ssm.jc_size()));
    CUDA_CHECK(hipMalloc(&ir_d, sizeof(int64_t) * ssm.ir_size()));
    CUDA_CHECK(hipMalloc(&vals_d, sizeof(float) * ssm.nnz()));

    // Convert from default Matlab types
    std::vector<int64_t> jc_64i(ssm.jc_size());
    for (int i = 0; i < ssm.jc_size(); i++)
    {
        jc_64i[i] = static_cast<int64_t>(ssm.jc()[i]);
    }
    CUDA_CHECK(hipMemcpy(jc_d, jc_64i.data(), sizeof(int64_t) * jc_64i.size(), hipMemcpyHostToDevice));

    std::vector<int64_t> ir_64i(ssm.ir_size());
    for (int i = 0; i < ssm.ir_size(); i++)
    {
        ir_64i[i] = static_cast<int64_t>(ssm.ir()[i]);
    }
    CUDA_CHECK(hipMemcpy(ir_d, ir_64i.data(), sizeof(int64_t) * ir_64i.size(), hipMemcpyHostToDevice));

    std::vector<float> nonzeros_32f(ssm.nnz());
    for (int i = 0; i < ssm.nnz(); i++)
    {
        nonzeros_32f[i] = static_cast<float>(ssm.data()[i]);
    }
    CUDA_CHECK(hipMemcpy(vals_d, nonzeros_32f.data(), sizeof(float) * nonzeros_32f.size(), hipMemcpyHostToDevice));

    hipsparseSpMatDescr_t A;
    CUSPARSE_CHECK(hipsparseCreateCsr(
        &A, ssm.rows(), ssm.cols(), ssm.nnz(),
        jc_d, ir_d, vals_d, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    const int n = ssm.rows();

    hipsparseDnMatDescr_t X;
    float *d_X = nullptr;
    CUDA_CHECK(hipMalloc(&d_X, sizeof(float) * n * s));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&X, n, s, n, d_X, HIP_R_32F, HIPSPARSE_ORDER_COL));

    hipsparseDnMatDescr_t B;
    float *d_B = nullptr;
    CUDA_CHECK(hipMalloc(&d_B, sizeof(float) * n * s));

    constexpr int block_size = 256;
    const size_t num_elements = n * s;
    const size_t num_blocks = (num_elements + block_size - 1) / block_size;
    set_val<<<num_blocks, block_size>>>(d_B, 1, num_elements);

    CUSPARSE_CHECK(hipsparseCreateDnMat(&B, n, s, n, d_B, HIP_R_32F, HIPSPARSE_ORDER_COL));

    constexpr float tolerance = std::numeric_limits<float>::epsilon();
    const int max_iterations = n;

    int iterations = 0;
    std::cerr << "TODO: Implement preconditioned DR-BCG" << std::endl;

    std::cout << iterations;

    return 0;
}
