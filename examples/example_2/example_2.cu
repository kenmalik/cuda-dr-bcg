#include "hip/hip_runtime.h"
#include <tuple>
#include <iostream>
#include <vector>
#include <string>

#include <suitesparse_matrix.h>

#include "dr_bcg/dr_bcg.h"
#include "dr_bcg/helper.h"

__global__ void set_val(float *d_A, float val, size_t num_elements)
{
    const int idx = blockIdx.x * blockDim.y + threadIdx.x;
    if (idx < num_elements)
    {
        d_A[idx] = val;
    }
}

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        std::cerr << "Usage: ./example_2 [.mat file]" << std::endl;
        return 1;
    }

    hipsolverHandle_t cusolverH;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    hipsolverDnParams_t cusolverP;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverP));

    hipblasHandle_t cublasH;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    hipsparseHandle_t cusparseH = NULL;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseH));

    const std::string matrix_file = argv[1];
    SuiteSparseMatrix ssm(matrix_file);

    int64_t *jc_d = nullptr;
    int64_t *ir_d = nullptr;
    float *vals_d = nullptr;

    float *x_d = nullptr;
    CUDA_CHECK(hipMalloc(&x_d, sizeof(float) * ssm.rows()));

    float *b_d = nullptr;
    std::vector<float> b_h(ssm.rows(), 1);
    CUDA_CHECK(hipMalloc(&b_d, sizeof(float) * b_h.size()));
    CUDA_CHECK(hipMemcpy(b_d, b_h.data(), sizeof(float) * b_h.size(), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&jc_d, sizeof(int64_t) * ssm.jc_size()));
    CUDA_CHECK(hipMalloc(&ir_d, sizeof(int64_t) * ssm.ir_size()));
    CUDA_CHECK(hipMalloc(&vals_d, sizeof(float) * ssm.nnz()));

    // Convert from default Matlab types
    std::vector<int64_t> jc_64i(ssm.jc_size());
    for (int i = 0; i < ssm.jc_size(); i++)
    {
        jc_64i[i] = static_cast<int64_t>(ssm.jc()[i]);
    }
    CUDA_CHECK(hipMemcpy(jc_d, jc_64i.data(), sizeof(int64_t) * jc_64i.size(), hipMemcpyHostToDevice));

    std::vector<int64_t> ir_64i(ssm.ir_size());
    for (int i = 0; i < ssm.ir_size(); i++)
    {
        ir_64i[i] = static_cast<int64_t>(ssm.ir()[i]);
    }
    CUDA_CHECK(hipMemcpy(ir_d, ir_64i.data(), sizeof(int64_t) * ir_64i.size(), hipMemcpyHostToDevice));

    std::vector<float> nonzeros_32f(ssm.nnz());
    for (int i = 0; i < ssm.nnz(); i++)
    {
        nonzeros_32f[i] = static_cast<float>(ssm.data()[i]);
    }
    CUDA_CHECK(hipMemcpy(vals_d, nonzeros_32f.data(), sizeof(float) * nonzeros_32f.size(), hipMemcpyHostToDevice));

    hipsparseSpMatDescr_t A;
    CUSPARSE_CHECK(hipsparseCreateCsr(
        &A, ssm.rows(), ssm.cols(), ssm.nnz(),
        jc_d, ir_d, vals_d, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    const int n = ssm.rows();
    const int s = 8;

    hipsparseDnMatDescr_t X;
    float *d_X = nullptr;
    CUDA_CHECK(hipMalloc(&d_X, sizeof(float) * n * s));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&X, n, s, n, d_X, HIP_R_32F, HIPSPARSE_ORDER_COL));

    hipsparseDnMatDescr_t B;
    float *d_B = nullptr;
    CUDA_CHECK(hipMalloc(&d_B, sizeof(float) * n * s));

    constexpr int block_size = 256;
    const size_t num_elements = n * s;
    const size_t num_blocks = (num_elements + block_size - 1) / block_size;
    set_val<<<num_blocks, block_size>>>(d_B, 1, num_elements);

    CUSPARSE_CHECK(hipsparseCreateDnMat(&B, n, s, n, d_B, HIP_R_32F, HIPSPARSE_ORDER_COL));

    constexpr float tolerance = 0.1;
    constexpr int max_iterations = 1000;

    int iterations = 0;
    std::cout << "Running" << std::endl;
    dr_bcg::dr_bcg(cusolverH, cusolverP, cublasH, cusparseH, A, X, B, tolerance, max_iterations, &iterations);

    std::cout << "Iterations: " << iterations << std::endl;

    return 0;
}
