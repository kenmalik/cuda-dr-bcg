#include "dr_bcg/device_buffer.h"
#include "dr_bcg/helper.h"

/**
 * @brief Constructor. Allocates all device buffers.
 * @param n n dimension
 * @param s s dimension
 */
DeviceBuffer::DeviceBuffer(int n, int s)
{
    allocate(n, s);
}

/**
 * @brief Destructor. Frees all allocated device memory.
 */
DeviceBuffer::~DeviceBuffer()
{
    deallocate();
}

/**
 * @brief Allocates device memory for all buffers.
 * @param n n dimension
 * @param s s dimension
 */
void DeviceBuffer::allocate(int n, int s)
{
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&w), sizeof(float) * n * s));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&sigma), sizeof(float) * s * s));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&(this->s)), sizeof(float) * n * s));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&xi), sizeof(float) * s * s));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&zeta), sizeof(float) * s * s));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&temp), sizeof(float) * n * s));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&residual), sizeof(float) * n));
}

/**
 * @brief Deallocates all device memory.
 */
void DeviceBuffer::deallocate()
{
    CUDA_CHECK(hipFree(w));
    CUDA_CHECK(hipFree(sigma));
    CUDA_CHECK(hipFree(s));
    CUDA_CHECK(hipFree(xi));
    CUDA_CHECK(hipFree(zeta));
    CUDA_CHECK(hipFree(temp));
    CUDA_CHECK(hipFree(residual));
}