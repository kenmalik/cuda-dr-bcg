#include "dr_bcg/device_buffer.h"
#include "dr_bcg/helper.h"

/**
 * @brief Constructor. Allocates all device buffers.
 * @param m m dimension
 * @param n n dimension
 */
DeviceBuffer::DeviceBuffer(int m, int n)
{
    allocate(m, n);
}

/**
 * @brief Destructor. Frees all allocated device memory.
 */
DeviceBuffer::~DeviceBuffer()
{
    deallocate();
}

/**
 * @brief Allocates device memory for all buffers.
 * @param m m dimension
 * @param n n dimension
 */
void DeviceBuffer::allocate(int m, int n)
{
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&w), sizeof(float) * m * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&sigma), sizeof(float) * n * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&s), sizeof(float) * m * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&xi), sizeof(float) * n * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&zeta), sizeof(float) * n * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&temp), sizeof(float) * m * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&residual), sizeof(float) * m));
}

/**
 * @brief Deallocates all device memory.
 */
void DeviceBuffer::deallocate()
{
    CUDA_CHECK(hipFree(w));
    CUDA_CHECK(hipFree(sigma));
    CUDA_CHECK(hipFree(s));
    CUDA_CHECK(hipFree(xi));
    CUDA_CHECK(hipFree(zeta));
    CUDA_CHECK(hipFree(temp));
    CUDA_CHECK(hipFree(residual));
}