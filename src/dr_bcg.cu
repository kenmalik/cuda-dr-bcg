#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <tuple>
#include <string>

#include <nvtx3/nvtx3.hpp>

#include "dr_bcg/dr_bcg.h"
#include "dr_bcg/helper.h"

/**
 * @brief Convenience wrapper for DR-BCG solver routine.
 *
 * Solves the block linear system AX = B using the DR-BCG algorithm, taking vectors and allocating device memory as required.
 *
 * @param A Host vector representing input matrix A (n x n)
 * @param X Host vector representing initial guess X (n x s)
 * @param B Host vector representing right-hand side B (n x s)
 * @param n n dimension
 * @param s s dimension
 * @param tolerance Relative residual tolerance for convergence
 * @param max_iterations Maximum number of iterations
 * @return Tuple containing the solution X (as a std::vector<float>) and the number of iterations performed
 */
std::tuple<std::vector<float>, int> dr_bcg::dr_bcg(
    const std::vector<float> &A,
    const std::vector<float> &X,
    const std::vector<float> &B,
    const int n,
    const int s,
    const float tolerance,
    const int max_iterations)
{
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnParams_t cusolverParams = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverParams));

    std::vector<float> X_final(n * s);
    int iterations = 0;

    float *d_A = nullptr;
    float *d_X = nullptr;
    float *d_B = nullptr;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * n * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_X), sizeof(float) * n * s));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * n * s));

    CUDA_CHECK(hipMemcpy(d_A, A.data(), sizeof(float) * n * n, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_X, X.data(), sizeof(float) * n * s, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B.data(), sizeof(float) * n * s, hipMemcpyHostToDevice));

    CUSOLVER_CHECK(dr_bcg(cusolverH, cusolverParams, cublasH, n, s, d_A, d_X, d_B, tolerance, max_iterations, &iterations));

    CUDA_CHECK(hipMemcpy(X_final.data(), d_X, sizeof(float) * n * s, hipMemcpyDeviceToHost));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUSOLVER_CHECK(hipsolverDnDestroyParams(cusolverParams));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_B));

    return {X_final, iterations};
}

/**
 * @brief Main DR-BCG solver routine.
 *
 * Solves the block linear system AX = B using the DR-BCG algorithm on device pointers.
 *
 * @param cusolverH cuSOLVER handle
 * @param cusolverParams cuSOLVER params
 * @param cublasH cuBLAS handle
 * @param n n dimension
 * @param s s dimension
 * @param A Device pointer to input matrix A (n x n)
 * @param X Device pointer to initial guess X (n x s), overwritten with solution
 * @param B Device pointer to right-hand side B (n x s)
 * @param tolerance Relative residual tolerance for convergence
 * @param max_iterations Maximum number of iterations
 * @param iterations Pointer to int, overwritten with number of iterations performed
 * @return cuSOLVER status
 */
hipsolverStatus_t dr_bcg::dr_bcg(
    hipsolverHandle_t cusolverH,
    hipsolverDnParams_t cusolverParams,
    hipblasHandle_t cublasH,
    int n,
    int s,
    const float *A,
    float *X,
    const float *B,
    float tolerance,
    int max_iterations,
    int *iterations)
{
    NVTX3_FUNC_RANGE();

    DeviceBuffer d(n, s);

#ifdef USE_TENSOR_CORES
    CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TF32_TENSOR_OP_MATH));
#endif

    // We don't include d_R in device buffers because it is only used once at the beginning
    // of the algorithm.
    float *d_R;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(float) * n * s));

    // R = B - AX
    get_R(cublasH, d_R, n, s, A, X, B);

#ifdef USE_THIN_QR
    thin_qr(cusolverH, cusolverParams, cublasH, d.w, d.sigma, n, s, d_R);
#else
    qr_factorization(cusolverH, cusolverParams, d.w, d.sigma, n, s, d_R);
#endif

    CUDA_CHECK(hipFree(d_R)); // Never used later

    // s = w
    CUDA_CHECK(hipMemcpy(d.s, d.w, sizeof(float) * n * s, hipMemcpyDeviceToDevice));

    float B1_norm;
    CUBLAS_CHECK(hipblasSnrm2(cublasH, n, B, 1, &B1_norm));

    *iterations = 0;
    while (*iterations < max_iterations)
    {
        nvtx3::scoped_range loop{"iteration"};

        (*iterations)++;

        // xi = (s' * A * s)^-1
        get_xi(cusolverH, cusolverParams, cublasH, n, s, d, A);

        // X = X + s * xi * sigma
        get_next_X(cublasH, n, s, d.s, d.xi, d.temp, d.sigma, X);

        // norm(B(:,1) - A * X(:,1)) / norm(B(:,1))
        float relative_residual_norm;
        residual(cublasH, d.residual, B, n, A, X);

        CUBLAS_CHECK(hipblasSnrm2(cublasH, n, d.residual, 1, &relative_residual_norm));
        relative_residual_norm /= B1_norm;

        if (relative_residual_norm < tolerance)
        {
            break;
        }
        else
        {
            nvtx3::scoped_range new_s_and_sigma{"get_new_s_and_sigma"};

            get_w_zeta(cusolverH, cusolverParams, cublasH, n, s, d, A);

            get_s(cublasH, n, s, d);

            get_sigma(cublasH, s, d);
        }
    }

    return HIPSOLVER_STATUS_SUCCESS;
}

void dr_bcg::get_xi(
    hipsolverHandle_t &cusolverH, hipsolverDnParams_t &cusolverParams, hipblasHandle_t &cublasH,
    const int n, const int s, DeviceBuffer &d, const float *d_A)
{
    NVTX3_FUNC_RANGE();

    quadratic_form(cublasH, n, s, d.s, d_A, d.temp, d.xi);
    invert_square_matrix(cusolverH, cusolverParams, d.xi, s);
}

void dr_bcg::get_sigma(hipblasHandle_t cublasH, int s, DeviceBuffer &d)
{
    NVTX3_FUNC_RANGE();

    // sigma = zeta * sigma
    constexpr float sgemm_alpha = 1;
    constexpr float sgemm_beta = 0;
    CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, s, s, s,
                                &sgemm_alpha, d.zeta, s, d.sigma, s,
                                &sgemm_beta, d.temp, s));
    CUDA_CHECK(hipMemcpy(d.sigma, d.temp, sizeof(float) * s * s, hipMemcpyDeviceToDevice));
}

void dr_bcg::get_s(hipblasHandle_t cublasH, const int n, const int s, DeviceBuffer &d)
{
    NVTX3_FUNC_RANGE();

    // temp = s * zeta'
    constexpr float strmm_alpha = 1;
    CUBLAS_CHECK(hipblasStrmm(cublasH, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER,
                                HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, n, s,
                                &strmm_alpha, d.zeta, s, d.s, n, d.temp, n));

    // s = w + temp
    constexpr float sgeam_alpha = 1;
    constexpr float sgeam_beta = 1;
    CUBLAS_CHECK(hipblasSgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, s,
                             &sgeam_alpha, d.w, n, &sgeam_beta, d.temp, n, d.s, n));
}

void dr_bcg::get_w_zeta(
    hipsolverHandle_t &cusolverH, hipsolverDnParams_t &cusolverParams, hipblasHandle_t &cublasH,
    const int n, const int s, DeviceBuffer &d, const float *d_A)
{
    NVTX3_FUNC_RANGE();

    // temp = A * s
    constexpr float alpha_1 = 1;
    constexpr float beta_1 = 0;
    CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, s, n,
                                &alpha_1, d_A, n, d.s, n,
                                &beta_1, d.temp, n));

    // w - temp * xi
    constexpr float alpha_2 = -1;
    constexpr float beta_2 = 1;
    CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, s, s,
                                &alpha_2, d.temp, n, d.xi, s,
                                &beta_2, d.w, n));

#ifdef USE_THIN_QR
    thin_qr(cusolverH, cusolverParams, cublasH, d.w, d.zeta, n, s, d.w);
#else
    qr_factorization(cusolverH, cusolverParams, d.w, d.zeta, n, s, d.w);
#endif
}

/**
 * @brief Calculates residual with the following formula: B^(1) - A * X^(1)
 *
 * @param cublasH cuBLAS handle
 * @param d_residual Device workspace for calculation. Result is overwritten to pointed location.
 * @param B Pointer to host memory B
 * @param n The n-value (represents dimensions of square matrix A and length of X and B)
 * @param d_A Pointer to device memory A
 * @param d_X Pointer to device memory X
 */
void dr_bcg::residual(hipblasHandle_t &cublasH, float *d_residual, const float *B, const int n, const float *d_A, const float *d_X)
{
    NVTX3_FUNC_RANGE();

    CUDA_CHECK(hipMemcpy(d_residual, B, sizeof(float) * n, hipMemcpyDeviceToDevice));

    constexpr float alpha = -1;
    constexpr float beta = 1;
    CUBLAS_CHECK(hipblasSgemv(
        cublasH, HIPBLAS_OP_N, n, n,
        &alpha, d_A, n, d_X, 1,
        &beta, d_residual, 1));
}

/**
 * @brief Calculates next X guess with the following formula: X_{i+1} = X_{i} + s * xi * sigma
 *
 * @param cublasH cuBLAS handle
 * @param n n dimension
 * @param s s dimension
 * @param d_s Device pointer to s (n x s)
 * @param d_xi Device pointer to xi (s x s)
 * @param d_temp Device pointer to temporary buffer (n x s)
 * @param d_sigma Device pointer to sigma (s x s)
 * @param d_X Device pointer to X (n x s). Result is overwritten to pointed location.
 */
void dr_bcg::get_next_X(hipblasHandle_t &cublasH, const int n, const int s, const float *d_s, const float *d_xi, float *d_temp, const float *d_sigma, float *d_X)
{
    NVTX3_FUNC_RANGE();

    constexpr float alpha = 1;
    constexpr float beta = 1;
    CUBLAS_CHECK(hipblasStrmm(cublasH, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                                s, s, &alpha, d_sigma, s, d_xi, s, d_temp, s));
    CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, s, s,
                                &alpha, d_s, n, d_temp, s,
                                &beta, d_X, n));
}

/**
 * @brief Compute y = x^T * A * x
 *
 * @param cublasH cuBLAS handle
 * @param m m dimension
 * @param n n dimension
 * @param d_x Device pointer to x (n x m)
 * @param d_A Device pointer to A (m x m)
 * @param d_work Device pointer to workspace
 * @param d_y Device pointer to result y (n x n)
 */
void dr_bcg::quadratic_form(hipblasHandle_t &cublasH, const int m, const int n,
                            const float *d_x, const float *d_A,
                            float *d_work, float *d_y)
{
    NVTX3_FUNC_RANGE();

    constexpr float alpha = 1;
    constexpr float beta = 0;
    CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, m,
                                &alpha, d_x, m, d_A, m,
                                &beta, d_work, n));
    CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, m,
                                &alpha, d_work, n, d_x, m,
                                &beta, d_y, n));
}

/**
 * @brief Computes R = B - AX as GEMM: R = -1.0 * AX + R where R initially contains B.
 *
 * @param cublasH cuBLAS handle
 * @param d_R Device pointer to result R (n x s)
 * @param n n dimension
 * @param s s dimension
 * @param A Host pointer to A (n x n)
 * @param X Host pointer to X (n x s)
 * @param B Host pointer to B (n x s)
 */
void dr_bcg::get_R(hipblasHandle_t &cublasH, float *R, const int n, const int s, const float *A, const float *X, const float *B)
{
    NVTX3_FUNC_RANGE();

    constexpr float alpha = -1;
    constexpr float beta = 1;

    CUDA_CHECK(hipMemcpy(R, B, sizeof(float) * n * s, hipMemcpyDeviceToDevice));

    CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                n, s, n,
                                &alpha, A, n, X, n,
                                &beta, R, n));
}

hipsolverStatus_t dr_bcg::dr_bcg(
    hipsolverHandle_t cusolverH,
    hipsolverDnParams_t cusolverParams,
    hipblasHandle_t cublasH,
    hipsparseHandle_t cusparseH,
    hipsparseSpMatDescr_t &A,
    hipsparseDnMatDescr_t &X,
    hipsparseDnMatDescr_t &B,
    float tolerance,
    int max_iterations,
    int *iterations)
{
    NVTX3_FUNC_RANGE();

    int64_t n = 0;
    int64_t s = 0;
    int64_t ld_X = 0;
    float *d_X = nullptr;
    hipDataType X_dtype;
    hipsparseOrder_t X_order;
    CUSPARSE_CHECK(hipsparseDnMatGet(X, &n, &s, &ld_X, reinterpret_cast<void **>(&d_X), &X_dtype, &X_order));

    float *d_B = nullptr;
    CUSPARSE_CHECK(hipsparseDnMatGetValues(B, reinterpret_cast<void **>(&d_B)));

    DeviceBuffer d(n, s);

#ifdef USE_TENSOR_CORES
    CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TF32_TENSOR_OP_MATH));
#endif

    // We don't include d_R in device buffers because it is only used once at the beginning
    // of the algorithm.
    hipsparseDnMatDescr_t R;
    float *d_R;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(float) * n * s));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&R, n, s, n, d_R, HIP_R_32F, HIPSPARSE_ORDER_COL));

    // R = B - AX
    get_R(cublasH, cusparseH, R, A, X, B);

#ifdef USE_THIN_QR
    thin_qr(cusolverH, cusolverParams, cublasH, d.w, d.sigma, n, s, d_R);
#else
    qr_factorization(cusolverH, cusolverParams, d.w, d.sigma, n, s, d_R);
#endif

    // R never used later
    CUDA_CHECK(hipFree(d_R));
    CUSPARSE_CHECK(hipsparseDestroyDnMat(R));

    // s = w
    CUDA_CHECK(hipMemcpy(d.s, d.w, sizeof(float) * n * s, hipMemcpyDeviceToDevice));

    float B1_norm;
    constexpr int stride = 1;
    CUBLAS_CHECK(hipblasSnrm2(cublasH, n, d_B, stride, &B1_norm));

    hipsparseDnVecDescr_t r;
    CUSPARSE_CHECK(hipsparseCreateDnVec(&r, n, d.residual, HIP_R_32F));

    int i = 0;
    while (i < max_iterations)
    {
        nvtx3::scoped_range loop{"iteration"};
        ++i;

        // xi = (s' * A * s)^-1
        get_xi(cublasH, cusolverH, cusolverParams, cusparseH, A, n, s, d);

        // X = X + s * xi * sigma
        get_next_X(cublasH, n, s, d.s, d.xi, d.temp, d.sigma, d_X);

        // norm(B(:,1) - A * X(:,1)) / norm(B(:,1))
        float relative_residual_norm;
        residual(cusparseH, r, d_B, A, X);

        CUBLAS_CHECK(hipblasSnrm2(cublasH, n, d.residual, stride, &relative_residual_norm));
        relative_residual_norm /= B1_norm;

        if (relative_residual_norm < tolerance)
        {
            break;
        }
        else
        {
            nvtx3::scoped_range new_s_and_sigma{"get_new_s_and_sigma"};

            get_w_zeta(cusolverH, cusolverParams, cublasH, cusparseH, n, s, d, A);

            get_s(cublasH, n, s, d);

            get_sigma(cublasH, s, d);
        }
    }

    if (iterations)
    {
        *iterations = i;
    }

    return HIPSOLVER_STATUS_SUCCESS;
}

void dr_bcg::get_R(
    hipblasHandle_t &cublasH,
    hipsparseHandle_t &cusparseH,
    hipsparseDnMatDescr_t &R,
    hipsparseSpMatDescr_t &A,
    hipsparseDnMatDescr_t &X,
    hipsparseDnMatDescr_t &B)
{
    NVTX3_FUNC_RANGE();

    constexpr float alpha = -1;
    constexpr float beta = 1;

    float *d_R = nullptr;
    int64_t n = 0;
    int64_t s = 0;
    int64_t ld = 0;
    hipDataType type;
    hipsparseOrder_t order;
    CUSPARSE_CHECK(hipsparseDnMatGet(R, &n, &s, &ld, reinterpret_cast<void **>(&d_R), &type, &order));
    float *d_B = nullptr;
    CUSPARSE_CHECK(hipsparseDnMatGetValues(B, reinterpret_cast<void **>(&d_B)));

    CUDA_CHECK(hipMemcpy(d_R, d_B, sizeof(float) * n * s, hipMemcpyDeviceToDevice));

    constexpr hipsparseOperation_t transpose = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    constexpr hipDataType compute_type = HIP_R_32F;
    constexpr hipsparseSpMMAlg_t algorithm_type = HIPSPARSE_SPMM_ALG_DEFAULT;

    void *buffer = nullptr;
    size_t buffer_size = 0;
    CUSPARSE_CHECK(hipsparseSpMM_bufferSize(
        cusparseH, transpose, transpose,
        &alpha, A, X, &beta, R,
        compute_type, algorithm_type, &buffer_size));

    if (buffer_size > 0)
    {
        CUDA_CHECK(hipMalloc(&buffer, buffer_size));
    }

    CUSPARSE_CHECK(hipsparseSpMM(
        cusparseH, transpose, transpose,
        &alpha, A, X, &beta, R,
        compute_type, algorithm_type, buffer));

    if (buffer)
    {
        CUDA_CHECK(hipFree(buffer));
    }
}

void dr_bcg::get_xi(
    hipblasHandle_t &cublasH,
    hipsolverHandle_t &cusolverH,
    hipsolverDnParams_t &cusolverParams,
    hipsparseHandle_t &cusparseH,
    hipsparseSpMatDescr_t &A,
    const int n,
    const int s,
    DeviceBuffer &d)
{
    NVTX3_FUNC_RANGE();

    hipsparseDnMatDescr_t s_descr;
    CUSPARSE_CHECK(hipsparseCreateDnMat(&s_descr, n, s, n, reinterpret_cast<void *>(d.s), HIP_R_32F, HIPSPARSE_ORDER_COL));

    {
        nvtx3::scoped_range quadform{"get_xi.quadratic_form"};
        quadratic_form(cublasH, cusparseH, n, s, s_descr, A, d.temp, d.xi);
    }

    {
        nvtx3::scoped_range invert{"get_xi.invert_square_matrix"};
        invert_square_matrix(cusolverH, cusolverParams, d.xi, s);
    }
}

void dr_bcg::quadratic_form(
    hipblasHandle_t &cublasH,
    hipsparseHandle_t &cusparseH,
    const int n,
    const int s,
    hipsparseDnMatDescr_t &X,
    hipsparseSpMatDescr_t &A,
    float *d_work,
    float *d_y)
{
    NVTX3_FUNC_RANGE();

    constexpr float alpha = 1;
    constexpr float beta = 0;

    constexpr hipsparseOperation_t X_transpose = HIPSPARSE_OPERATION_TRANSPOSE;
    constexpr hipsparseOperation_t A_transpose = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    constexpr hipDataType compute_type = HIP_R_32F;
    constexpr hipsparseSpMMAlg_t algorithm_type = HIPSPARSE_SPMM_ALG_DEFAULT;

    // Ax
    hipsparseDnMatDescr_t work_descr;
    CUSPARSE_CHECK(hipsparseCreateDnMat(&work_descr, n, s, n, reinterpret_cast<void *>(d_work), HIP_R_32F, HIPSPARSE_ORDER_COL));

    void *buffer = nullptr;
    size_t buffer_size = 0;
    CUSPARSE_CHECK(hipsparseSpMM_bufferSize(
        cusparseH, X_transpose, A_transpose,
        &alpha, A, X, &beta, work_descr,
        compute_type, algorithm_type, &buffer_size));

    if (buffer_size > 0)
    {
        CUDA_CHECK(hipMalloc(&buffer, buffer_size));
    }

    CUSPARSE_CHECK(hipsparseSpMM(
        cusparseH, X_transpose, A_transpose,
        &alpha, A, X, &beta, work_descr,
        compute_type, algorithm_type, buffer));

    if (buffer)
    {
        CUDA_CHECK(hipFree(buffer));
    }

    CUSPARSE_CHECK(hipsparseDestroyDnMat(work_descr));

    // x^TAx
    float *d_X = nullptr;
    CUSPARSE_CHECK(hipsparseDnMatGetValues(X, reinterpret_cast<void **>(&d_X)));

    CUBLAS_CHECK(hipblasSgemm(
        cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, s, s, n,
        &alpha, d_X, n, d_work, n,
        &beta, d_y, s));
}

void dr_bcg::residual(
    hipsparseHandle_t &cusparseH,
    hipsparseDnVecDescr_t &residual,
    const float *B,
    hipsparseSpMatDescr_t &A,
    hipsparseDnMatDescr_t &X)
{
    NVTX3_FUNC_RANGE();

    int64_t n = 0;
    float *d_residual = nullptr;
    hipDataType residual_dtype;
    CUSPARSE_CHECK(hipsparseDnVecGet(residual, &n, reinterpret_cast<void **>(&d_residual), &residual_dtype));

    CUDA_CHECK(hipMemcpy(d_residual, B, sizeof(float) * n, hipMemcpyDeviceToDevice));

    float *d_X = nullptr;
    CUSPARSE_CHECK(hipsparseDnMatGetValues(X, reinterpret_cast<void **>(&d_X)));
    hipsparseDnVecDescr_t X_1;
    CUSPARSE_CHECK(hipsparseCreateDnVec(&X_1, n, d_X, HIP_R_32F));

    constexpr float alpha = -1;
    constexpr float beta = 1;

    void *buffer = nullptr;
    size_t buffer_size = 0;
    CUSPARSE_CHECK(hipsparseSpMV_bufferSize(
        cusparseH, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, A, X_1, &beta, residual,
        HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &buffer_size));

    if (buffer_size > 0)
    {
        CUDA_CHECK(hipMalloc(&buffer, buffer_size));
    }

    CUSPARSE_CHECK(hipsparseSpMV(
        cusparseH, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, A, X_1, &beta, residual,
        HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    if (buffer)
    {
        CUDA_CHECK(hipFree(buffer));
    }

    CUSPARSE_CHECK(hipsparseDestroyDnVec(X_1));
}

void dr_bcg::get_w_zeta(
    hipsolverHandle_t &cusolverH,
    hipsolverDnParams_t &cusolverParams,
    hipblasHandle_t &cublasH,
    hipsparseHandle_t &cusparseH,
    const int n,
    const int s,
    DeviceBuffer &d,
    hipsparseSpMatDescr_t &A)
{
    NVTX3_FUNC_RANGE();

    constexpr hipsparseOperation_t transpose = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    constexpr hipDataType compute_type = HIP_R_32F;
    constexpr hipsparseSpMMAlg_t mm_type = HIPSPARSE_SPMM_ALG_DEFAULT;

    void *buffer = nullptr;
    size_t buffer_size = 0;

    hipsparseDnMatDescr_t s_desc;
    CUSPARSE_CHECK(hipsparseCreateDnMat(&s_desc, n, s, n, d.s, HIP_R_32F, HIPSPARSE_ORDER_COL));

    hipsparseDnMatDescr_t work;
    CUSPARSE_CHECK(hipsparseCreateDnMat(&work, n, s, n, d.temp, HIP_R_32F, HIPSPARSE_ORDER_COL));

    {
        // temp = A * s
        nvtx3::scoped_range SpMM{"get_w_zeta.SpMM"};
        constexpr float alpha_1 = 1;
        constexpr float beta_1 = 0;
        CUSPARSE_CHECK(hipsparseSpMM_bufferSize(
            cusparseH, transpose, transpose,
            &alpha_1, A, s_desc, &beta_1, work,
            compute_type, mm_type, &buffer_size));

        if (buffer_size > 0)
        {
            CUDA_CHECK(hipMalloc(&buffer, buffer_size));
        }

        CUSPARSE_CHECK(hipsparseSpMM(
            cusparseH, transpose, transpose,
            &alpha_1, A, s_desc, &beta_1, work,
            compute_type, mm_type, buffer));

        if (buffer)
        {
            CUDA_CHECK(hipFree(buffer));
        }
    }

    {
        nvtx3::scoped_range Sgemm{"get_w_zeta.Sgemm"};
        // w - temp * xi
        constexpr float alpha_2 = -1;
        constexpr float beta_2 = 1;
        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, s, s,
                                    &alpha_2, d.temp, n, d.xi, s,
                                    &beta_2, d.w, n));
    }

    {
        nvtx3::scoped_range factorization{"get_w_zeta.factorization"};
#ifdef USE_THIN_QR
        thin_qr(cusolverH, cusolverParams, cublasH, d.w, d.zeta, n, s, d.w);
#else
        qr_factorization(cusolverH, cusolverParams, d.w, d.zeta, n, s, d.w);
#endif
    }
}