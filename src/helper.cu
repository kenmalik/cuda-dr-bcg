#include <iostream>
#include <vector>
#include <hipblas.h>
#include "dr_bcg/helper.h"

/// @brief Prints a matrix stored in column-major order
void print_matrix(const float *mat, const int rows, const int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%7.2f ", mat[j * rows + i]);
        }
        std::cout << std::endl;
    }
}

void print_device_matrix(const float *d_mat, const int rows, const int cols) {
    std::vector<float> h_mat(rows * cols);
    CUDA_CHECK(hipMemcpy(h_mat.data(), d_mat, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
    print_matrix(h_mat.data(), rows, cols);
}

void fill_random(float *mat, const int rows, const int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            mat[i * cols + j] = std::rand() % 100 / 100.0;
        }
    }
}

void fill_spd(float *mat, const int n)
{
    fill_random(mat, n, n);

    hipblasHandle_t cublasH;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    float alpha = 1.0;
    float beta = 0.0;

    float *d_mat = nullptr;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_mat), sizeof(float) * n * n));
    CUDA_CHECK(hipMemcpy(d_mat, mat, sizeof(float) * n * n, hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasSgemm(
        cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, n,
        &alpha, d_mat, n, d_mat, n, &beta, d_mat, n));

    CUDA_CHECK(hipMemcpy(mat, d_mat, sizeof(float) * n * n, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_mat));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
}