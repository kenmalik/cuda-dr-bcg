#include <iostream>
#include <vector>
#include <hipblas.h>
#include "dr_bcg/helper.h"

/**
 * @brief Prints a matrix stored in column-major order.
 * 
 * @param mat Pointer to the matrix data (column-major)
 * @param rows Number of rows in the matrix
 * @param cols Number of columns in the matrix
 */
void print_matrix(const float *mat, const int rows, const int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%7.2f ", mat[j * rows + i]);
        }
        std::cout << std::endl;
    }
}

/**
 * @brief Prints a device matrix by copying it to host and calling print_matrix.
 * 
 * @param d_mat Device pointer to the matrix (column-major)
 * @param rows Number of rows in the matrix
 * @param cols Number of columns in the matrix
 */
void print_device_matrix(const float *d_mat, const int rows, const int cols) {
    std::vector<float> h_mat(rows * cols);
    CUDA_CHECK(hipMemcpy(h_mat.data(), d_mat, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
    print_matrix(h_mat.data(), rows, cols);
}

/**
 * @brief Fills a matrix with random values in the range [0, 1).
 * 
 * @param mat Pointer to the matrix data (host)
 * @param rows Number of rows in the matrix
 * @param cols Number of columns in the matrix
 */
void fill_random(float *mat, const int rows, const int cols)
{
    for (int j = 0; j < cols; j++)
    {
        for (int i = 0; i < rows; i++)
        {
            mat[j * rows + i] = std::rand() % 100 / 100.0;
        }
    }
}

/**
 * @brief Fills a matrix with random values and makes it symmetric positive definite (SPD).
 * 
 * The matrix is filled with random values, then multiplied by its transpose to ensure SPD.
 * 
 * @param mat Pointer to the matrix data (host)
 * @param n Matrix dimensions
 */
void fill_spd(float *mat, const int n)
{
    fill_random(mat, n, n);

    hipblasHandle_t cublasH;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    float alpha = 1.0 / n;
    float beta = 0.0;

    float *d_mat = nullptr;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_mat), sizeof(float) * n * n));
    CUDA_CHECK(hipMemcpy(d_mat, mat, sizeof(float) * n * n, hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasSgemm(
        cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, n,
        &alpha, d_mat, n, d_mat, n, &beta, d_mat, n));

    CUDA_CHECK(hipMemcpy(mat, d_mat, sizeof(float) * n * n, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_mat));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
}