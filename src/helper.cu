#include <iostream>
#include <hipblas.h>
#include "dr_bcg/helper.h"

/// @brief Prints a matrix stored in column-major order
void print_matrix(const float *mat, const int rows, const int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%6.3f ", mat[j * cols + i]);
        }
        std::cout << std::endl;
    }
}

void fill_random(float *mat, const int rows, const int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            mat[i * cols + j] = rand() % 100 / 100.0;
        }
    }
}

void fill_spd(float *mat, const int n) {
    fill_random(mat, n, n);
    
    hipblasHandle_t cublasH;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    float alpha = 1.0;
    float beta = 0.0;

    float *d_mat = nullptr;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_mat), sizeof(float) * n * n));
    CUDA_CHECK(hipMemcpy(d_mat, mat, sizeof(float) * n * n, hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasSgemm(
        cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, n,
        &alpha, d_mat, n, d_mat, n, &beta, d_mat, n
    ));

    CUDA_CHECK(hipMemcpy(mat, d_mat, sizeof(float) * n * n, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_mat));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
}