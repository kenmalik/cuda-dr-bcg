#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "dr_bcg/dr-bcg.h"
#include "dr_bcg/helper.h"

namespace dr_bcg
{
    /*
        * function [X_final, iterations] = DR_BCG(A, B, X, tol, maxit)
        *     iterations = 0;
        *     R = B - A * X;
        *     [w, sigma] = qr(R,'econ');
        *     s = w;

        *     for k = 1:maxit
        *         iterations = iterations + 1;
        *         xi = (s' * A * s)^-1;
        *         X = X + s * xi * sigma;
        *         if (norm(B(:,1) - A * X(:,1)) / norm(B(:,1))) < tol
        *             break
        *         else
        *             [w, zeta] = qr(w - A * s * xi,'econ');
        *             s = w + s * zeta';
        *             sigma = zeta * sigma;
        *         end
        *     end
        *     X_final = X;
        * end
        */
    int dr_bcg(
        float *A,
        const int n,
        const int m,
        const float *X,
        const float *B,
        const float tolerance,
        const int max_iterations)
    {
        int iterations = 0;

        hipblasHandle_t cublasH;
        CUBLAS_CHECK(hipblasCreate(&cublasH));

        std::vector<float> R(n * m);

        get_R(cublasH, R.data(), n, m, A, X, B);

        std::cout << "\nAfter R = B - AX\n"
                  << std::endl;
        std::cout << "A:" << std::endl;
        print_matrix(A, n, n);
        std::cout << "X:" << std::endl;
        print_matrix(X, n, m);
        std::cout << "B:" << std::endl;
        print_matrix(B, n, m);
        std::cout << "R:" << std::endl;
        print_matrix(R.data(), n, m);

        hipsolverHandle_t cusolverH = NULL;
        hipsolverDnParams_t cusolverParams = NULL;

        CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
        CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverParams));

        std::cout << "[INFO]Starting QR procedure [w, sigma] = qr(R)" << std::endl;
        std::vector<float> w(n * n);
        std::vector<float> sigma(n * n);
        qr_decomposition(cusolverH, cusolverParams, w.data(), sigma.data(), n, A);

        std::cout << "\nAfter [w, sigma] = qr(r)\n"
                  << std::endl;
        std::cout << "w:" << std::endl;
        print_matrix(w.data(), n, n);
        std::cout << "sigma:" << std::endl;
        print_matrix(sigma.data(), n, n);

        CUBLAS_CHECK(hipblasDestroy(cublasH));
        CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

        for (int k = 0; k < max_iterations; k++)
        {
            iterations++;
        }

        return iterations;
    }

    // R = B - AX as GEMM:
    // R = -1.0 * AX + R where R initially contains B
    void get_R(hipblasHandle_t &cublasH, float *h_R, const int n, const int m, const float *A, const float *X, const float *B)
    {
        constexpr float alpha = -1;
        constexpr float beta = 1;

        float *d_A = nullptr;
        float *d_X = nullptr;
        float *d_R = nullptr;

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * n * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_X), sizeof(float) * n * m));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(float) * n * m));

        CUDA_CHECK(hipMemcpy(d_A, A, sizeof(float) * n * n, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_X, X, sizeof(float) * n * m, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_R, B, sizeof(float) * n * m, hipMemcpyHostToDevice));

        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    n, m, n,
                                    &alpha, d_A, n, d_X, n,
                                    &beta, d_R, n));

        CUDA_CHECK(hipMemcpy(h_R, d_R, sizeof(float) * n * m, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_X));
        CUDA_CHECK(hipFree(d_R));
    }

    void qr_decomposition(hipsolverHandle_t &cusolverH, hipsolverDnParams_t &params, float *q, float *r, const int n, float *A)
    {
        std::vector<float> tau(n, 0);
        int info = 0;

        float *d_A = nullptr;
        float *d_tau = nullptr;
        int *d_info = nullptr;

        size_t lwork_geqrf_d = 0;
        void *d_work = nullptr;
        size_t lwork_geqrf_h = 0;
        void *h_work = nullptr;

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * n * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_tau), sizeof(float) * tau.size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

        CUDA_CHECK(hipMemcpy(d_A, A, sizeof(float) * n * n, hipMemcpyHostToDevice));

        CUSOLVER_CHECK(hipsolverDnXgeqrf_bufferSize(cusolverH, params, n, n, HIP_R_32F, d_A,
                                                   n, HIP_R_32F, d_tau,
                                                   HIP_R_32F, &lwork_geqrf_d,
                                                   &lwork_geqrf_h));

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), lwork_geqrf_d));

        if (0 < lwork_geqrf_h)
        {
            h_work = reinterpret_cast<void *>(malloc(lwork_geqrf_h));
            if (h_work == nullptr)
            {
                throw std::runtime_error("Error: h_work not allocated.");
            }
        }

        CUSOLVER_CHECK(hipsolverDnXgeqrf(cusolverH, params, n, n, HIP_R_32F, d_A,
                                        n, HIP_R_32F, d_tau,
                                        HIP_R_32F, d_work, lwork_geqrf_d, h_work,
                                        lwork_geqrf_h, d_info));

        // Copy R (stored in upper triangular)
        CUDA_CHECK(hipMemcpy(r, d_A, sizeof(float) * n * n, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipMemcpy(tau.data(), d_tau, sizeof(float) * tau.size(), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));

        if (0 > info)
        {
            std::printf("%d-th parameter is wrong \n", -info);
            exit(1);
        }

        CUDA_CHECK(hipMemcpy(A, d_A, sizeof(float) * n * n, hipMemcpyDeviceToHost));

        // Explicitly compute Q
        int lwork_orgqr = 0;
        CUSOLVER_CHECK(hipsolverDnSorgqr_bufferSize(cusolverH, n, n, n, d_A, n, d_tau, &lwork_orgqr));
        CUSOLVER_CHECK(hipsolverDnSorgqr(cusolverH, n, n, n, d_A, n, d_tau, reinterpret_cast<float *>(d_work), lwork_orgqr, d_info));

        // Copy Q
        CUDA_CHECK(hipMemcpy(q, d_A, sizeof(float) * n * n, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_info));
        CUDA_CHECK(hipFree(d_tau));
        CUDA_CHECK(hipFree(d_work));

        free(h_work);
    }
}
