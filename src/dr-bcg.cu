#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>

#include "dr_bcg/dr-bcg.h"
#include "dr_bcg/helper.h"

// Device pointers for reused device buffers
struct DeviceBuffer
{
    float *A = nullptr;
    float *X = nullptr;
    float *w = nullptr;
    float *sigma = nullptr;
    float *s = nullptr;
    float *xi = nullptr;
    float *zeta = nullptr;
    float *temp = nullptr;
    float *residual = nullptr;

    DeviceBuffer(int m, int n)
    {
        allocate(m, n);
    }

    ~DeviceBuffer()
    {
        deallocate();
    }

    void allocate(int m, int n)
    {
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&A), sizeof(float) * m * m));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&X), sizeof(float) * m * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&w), sizeof(float) * m * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&sigma), sizeof(float) * n * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&s), sizeof(float) * m * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&xi), sizeof(float) * n * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&zeta), sizeof(float) * n * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&temp), sizeof(float) * m * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&residual), sizeof(float) * m));
    }

    void deallocate()
    {
        CUDA_CHECK(hipFree(A));
        CUDA_CHECK(hipFree(X));
        CUDA_CHECK(hipFree(w));
        CUDA_CHECK(hipFree(sigma));
        CUDA_CHECK(hipFree(s));
        CUDA_CHECK(hipFree(xi));
        CUDA_CHECK(hipFree(zeta));
        CUDA_CHECK(hipFree(temp));
        CUDA_CHECK(hipFree(residual));
    }
};

__global__ void symmetrize_matrix(float *A, const int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < col && row < n && col < n)
    {
        A[col * n + row] = A[row * n + col];
    }
}

namespace dr_bcg
{
    int dr_bcg(
        const float *A,
        const int m,
        const int n,
        float *X,
        const float *B,
        const float tolerance,
        const int max_iterations)
    {
        hipblasHandle_t cublasH;
        CUBLAS_CHECK(hipblasCreate(&cublasH));

        hipsolverHandle_t cusolverH = NULL;
        hipsolverDnParams_t cusolverParams = NULL;
        CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
        CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverParams));

        DeviceBuffer d(m, n);
        CUDA_CHECK(hipMemcpy(d.A, A, sizeof(float) * m * m, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d.X, X, sizeof(float) * m * n, hipMemcpyHostToDevice));

        // We don't include d_R in device buffers because it is only used once at the beginning
        // of the algorithm.
        float *d_R;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(float) * m * n));

        // R = B - AX
        get_R(cublasH, d_R, m, n, A, X, B);

        // [w, sigma] = qr(R)
        qr_factorization(cusolverH, cusolverParams, d.w, d.sigma, m, n, d_R);

        CUDA_CHECK(hipFree(d_R)); // Never used later

        // s = w
        CUDA_CHECK(hipMemcpy(d.s, d.w, sizeof(float) * m * n, hipMemcpyDeviceToDevice));

        float B1_norm;
        float *d_B1 = nullptr;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B1), sizeof(float) * m));
        CUDA_CHECK(hipMemcpy(d_B1, B, sizeof(float) * m, hipMemcpyHostToDevice));
        CUBLAS_CHECK(hipblasSnrm2(cublasH, m, d_B1, 1, &B1_norm));
        CUDA_CHECK(hipFree(d_B1));

        int iterations;
        for (iterations = 1; iterations <= max_iterations; iterations++)
        {
            // xi = (s' * A * s)^-1
            quadratic_form(cublasH, m, n, d.s, d.A, d.temp, d.xi);
            invert_spd(cusolverH, cusolverParams, d.xi, n);

            // X = X + s * xi * sigma
            next_X(cublasH, m, n, d.s, d.xi, d.temp, d.sigma, d.X);

            // norm(B(:,1) - A * X(:,1)) / norm(B(:,1))
            float relative_residual_norm;
            residual(cublasH, d.residual, B, m, d.A, d.X);
            CUBLAS_CHECK(hipblasSnrm2(cublasH, m, d.residual, 1, &relative_residual_norm));
            relative_residual_norm /= B1_norm;

            if (relative_residual_norm < tolerance)
            {
                break;
            }
            else
            {
                // temp = A * s
                float alpha = 1;
                float beta = 0;
                CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, m,
                                            &alpha, d.A, m, d.s, m,
                                            &beta, d.temp, m));

                // w - temp * xi
                alpha = -1;
                beta = 1;
                CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, n,
                                            &alpha, d.temp, m, d.xi, n,
                                            &beta, d.w, m));

                qr_factorization(cusolverH, cusolverParams, d.w, d.zeta, m, n, d.w);

                // temp = s * zeta'
                alpha = 1;
                CUBLAS_CHECK(hipblasStrmm(cublasH, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER,
                                            HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, m, n,
                                            &alpha, d.zeta, n, d.s, m, d.temp, m));

                // s = w + temp
                beta = 1;
                CUBLAS_CHECK(hipblasSgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n,
                                         &alpha, d.w, m, &beta, d.temp, m, d.s, m));

                // sigma = zeta * sigma
                beta = 0;
                CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
                                            &alpha, d.zeta, n, d.sigma, n,
                                            &beta, d.temp, n));
                CUDA_CHECK(hipMemcpy(d.sigma, d.temp, sizeof(float) * n * n, hipMemcpyDeviceToDevice));
            }
        }

        CUDA_CHECK(hipMemcpy(X, d.X, sizeof(float) * m * n, hipMemcpyDeviceToHost));

        CUBLAS_CHECK(hipblasDestroy(cublasH));
        CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
        CUSOLVER_CHECK(hipsolverDnDestroyParams(cusolverParams));

        return iterations;
    }

    /// @brief Calculates residual with the following formula: B^(1) - A * X^(1)
    /// @param cublasH cuBLAS handle
    /// @param d_residual device workspace for calculation. Result is overwritten to pointed location
    /// @param B pointer to host memory B
    /// @param m the m-value (represents dimensions of square matrix A and length of X and B)
    /// @param d_A pointer to device memory A
    /// @param d_X pointer to device memory X
    void residual(hipblasHandle_t &cublasH, float *d_residual, const float *B, const int m, const float *d_A, const float *d_X)
    {
        CUDA_CHECK(hipMemcpy(d_residual, B, sizeof(float) * m, hipMemcpyHostToDevice));

        constexpr float alpha = -1;
        constexpr float beta = 1;
        CUBLAS_CHECK(hipblasSgemv(
            cublasH, HIPBLAS_OP_N, m, m,
            &alpha, d_A, m, d_X, 1,
            &beta, d_residual, 1));
    }

    /// @brief Calculates X_{i+1} = X_{i} + s * xi * sigma
    /// @param d_X (device memory pointer) X_{i}. Result is overwritten to pointed location
    void next_X(hipblasHandle_t &cublasH, const int m, const int n, const float *d_s, const float *d_xi, float *d_temp, const float *d_sigma, float *d_X)
    {
        constexpr float alpha = 1;
        constexpr float beta = 1;
        CUBLAS_CHECK(hipblasStrmm(cublasH, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                                    n, n, &alpha, d_sigma, n, d_xi, n, d_temp, n));
        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, n,
                                    &alpha, d_s, m, d_temp, n,
                                    &beta, d_X, m));
    }

    /// @brief Compute y = x^T * A * x
    void quadratic_form(hipblasHandle_t &cublasH, const int m, const int n,
                        const float *d_x, const float *d_A,
                        float *d_work, float *d_y)
    {
        constexpr float alpha = 1;
        constexpr float beta = 0;
        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, m,
                                    &alpha, d_x, m, d_A, m,
                                    &beta, d_work, n));
        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, m,
                                    &alpha, d_work, n, d_x, m,
                                    &beta, d_y, n));
    }

    // R = B - AX as GEMM:
    // R = -1.0 * AX + R where R initially contains B
    void get_R(hipblasHandle_t &cublasH, float *d_R, const int m, const int n, const float *A, const float *X, const float *B)
    {
        constexpr float alpha = -1;
        constexpr float beta = 1;

        float *d_A = nullptr;
        float *d_X = nullptr;

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * m * m));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_X), sizeof(float) * m * n));

        CUDA_CHECK(hipMemcpy(d_A, A, sizeof(float) * m * m, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_X, X, sizeof(float) * m * n, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_R, B, sizeof(float) * m * n, hipMemcpyHostToDevice));

        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    m, n, m,
                                    &alpha, d_A, m, d_X, m,
                                    &beta, d_R, m));

        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_X));
    }

    /// @brief Computes the QR factorization of matrix A
    /// @param cusolverH cuSOLVER handle
    /// @param params params for the cuSOLVER handle
    /// @param Q pointer to device memory to store Q result in
    /// @param R pointer to device memory to store R result in. Note that the lower triangular still contains householder vectors and must be handled accordingly
    /// (e.g. by using trmm in future multiplications using the R factor)
    /// @param m m-dimension (leading dimension) of A
    /// @param n n-dimension (second dimension) of A
    /// @param A the matrix to factorize
    void qr_factorization(hipsolverHandle_t &cusolverH, hipsolverDnParams_t &params, float *Q, float *R, const int m, const int n, const float *A)
    {
        int k = std::min(m, n);
        int info = 0;

        float *d_tau = nullptr;
        int *d_info = nullptr;

        size_t lwork_geqrf_d = 0;
        void *d_work = nullptr;
        size_t lwork_geqrf_h = 0;
        void *h_work = nullptr;

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_tau), sizeof(float) * k));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

        CUDA_CHECK(hipMemcpy(Q, A, sizeof(float) * m * n, hipMemcpyDeviceToDevice));

        CUSOLVER_CHECK(hipsolverDnXgeqrf_bufferSize(cusolverH, params, m, n, HIP_R_32F, Q,
                                                   m, HIP_R_32F, d_tau,
                                                   HIP_R_32F, &lwork_geqrf_d,
                                                   &lwork_geqrf_h));

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), lwork_geqrf_d));

        if (0 < lwork_geqrf_h)
        {
            h_work = reinterpret_cast<void *>(malloc(lwork_geqrf_h));
            if (h_work == nullptr)
            {
                throw std::runtime_error("Error: h_work not allocated.");
            }
        }

        CUSOLVER_CHECK(hipsolverDnXgeqrf(cusolverH, params, m, n, HIP_R_32F, Q,
                                        m, HIP_R_32F, d_tau,
                                        HIP_R_32F, d_work, lwork_geqrf_d, h_work,
                                        lwork_geqrf_h, d_info));
        if (h_work)
        {
            free(h_work); // No longer needed
        }

        const int max_R_col = std::min(m, n);
        for (int col = 0; col < max_R_col; col++)
        {
            CUDA_CHECK(hipMemcpy(R + col * n, Q + col * m, sizeof(float) * (col + 1), hipMemcpyDeviceToDevice));
        }

        CUDA_CHECK(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
        if (0 > info)
        {
            throw std::runtime_error(std::to_string(-info) + "-th parameter is wrong \n");
        }

        // Explicitly compute Q
        int lwork_orgqr = 0;
        CUSOLVER_CHECK(hipsolverDnSorgqr_bufferSize(cusolverH, m, n, k, Q, m, d_tau, &lwork_orgqr));
        if (lwork_orgqr > lwork_geqrf_d)
        {
            CUDA_CHECK(hipFree(d_work));
            CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(float) * lwork_orgqr));
        }

        CUSOLVER_CHECK(hipsolverDnSorgqr(cusolverH, m, n, k, Q, m, d_tau, reinterpret_cast<float *>(d_work), lwork_orgqr, d_info));

        CUDA_CHECK(hipFree(d_info));
        CUDA_CHECK(hipFree(d_tau));
        CUDA_CHECK(hipFree(d_work));
    }

    /// @brief Computes the inverse of a matrix using Cholesky factorization
    /// @param A (device memory pointer) the symmetric positive definite matrix to invert. Result is overwritten to pointed location.
    void invert_spd(hipsolverHandle_t &cusolverH, hipsolverDnParams_t &params, float *d_A, const int n)
    {
        size_t workspaceInBytesOnDevice = 0;
        void *d_work = nullptr;
        size_t workspaceInBytesOnHost = 0;
        void *h_work = nullptr;

        int info = 0;
        int *d_info = nullptr;

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

        CUSOLVER_CHECK(hipsolverDnXpotrf_bufferSize(cusolverH, params, HIPBLAS_FILL_MODE_LOWER,
                                                   n, HIP_R_32F, d_A, n, HIP_R_32F,
                                                   &workspaceInBytesOnDevice, &workspaceInBytesOnHost));

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice));
        if (0 < workspaceInBytesOnHost)
        {
            h_work = reinterpret_cast<void *>(malloc(sizeof(float) * workspaceInBytesOnHost));
            if (h_work == nullptr)
            {
                throw std::runtime_error("Error: h_work not allocated.");
            }
        }

        CUSOLVER_CHECK(hipsolverDnXpotrf(cusolverH, params, HIPBLAS_FILL_MODE_LOWER,
                                        n, HIP_R_32F, d_A, n,
                                        HIP_R_32F, d_work, workspaceInBytesOnDevice,
                                        h_work, workspaceInBytesOnHost, d_info));

        CUDA_CHECK(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
        if (0 > info)
        {
            throw std::runtime_error(std::to_string(-info) + "-th parameter is wrong \n");
        }
        CUDA_CHECK(hipFree(d_work));

        float *d_work_Spotri = nullptr;
        int lwork_Spotri = 0;
        info = 0;
        CUSOLVER_CHECK(hipsolverDnSpotri_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, n,
                                                   d_A, n, &lwork_Spotri));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work_Spotri), lwork_Spotri));
        CUSOLVER_CHECK(hipsolverDnSpotri(cusolverH, HIPBLAS_FILL_MODE_LOWER, n,
                                        d_A, n, d_work_Spotri, lwork_Spotri, d_info));

        CUDA_CHECK(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
        if (0 > info)
        {
            throw std::runtime_error(std::to_string(-info) + "-th parameter is wrong \n");
        }

        constexpr int block_n = 16;
        dim3 block_dim(block_n, block_n);
        dim3 grid_dim((n + block_n - 1) / block_n, (n + block_n - 1) / block_n);
        symmetrize_matrix<<<grid_dim, block_dim>>>(d_A, n);

        CUDA_CHECK(hipFree(d_work_Spotri));

        CUDA_CHECK(hipFree(d_info));
    }
}
