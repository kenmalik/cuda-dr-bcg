#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "dr_bcg/dr-bcg.h"
#include "dr_bcg/helper.h"

namespace dr_bcg
{
    int dr_bcg(
        float *A,
        const int m,
        const int n,
        const float *X,
        const float *B,
        const float tolerance,
        const int max_iterations)
    {
        int iterations = 0;

        hipblasHandle_t cublasH;
        CUBLAS_CHECK(hipblasCreate(&cublasH));

        // R = B - AX
        std::vector<float> R(m * n);
        get_R(cublasH, R.data(), m, n, A, X, B);

        std::cout << "\nAfter R = B - AX\n"
                  << std::endl;
        std::cout << "A:" << std::endl;
        print_matrix(A, m, m);
        std::cout << "X:" << std::endl;
        print_matrix(X, m, n);
        std::cout << "B:" << std::endl;
        print_matrix(B, m, n);
        std::cout << "R:" << std::endl;
        print_matrix(R.data(), m, n);

        hipsolverHandle_t cusolverH = NULL;
        hipsolverDnParams_t cusolverParams = NULL;

        CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
        CUSOLVER_CHECK(hipsolverDnCreateParams(&cusolverParams));

        // [w, sigma] = qr(R)
        std::vector<float> w(m * n);
        std::vector<float> sigma(n * n);
        qr_factorization(cusolverH, cusolverParams, w.data(), sigma.data(), m, n, R.data());

        std::cout << "\nAfter [w, sigma] = qr(R)\n"
                  << std::endl;
        std::cout << "w:" << std::endl;
        print_matrix(w.data(), m, n);
        std::cout << "sigma:" << std::endl;
        print_matrix(sigma.data(), n, n);

        CUBLAS_CHECK(hipblasDestroy(cublasH));
        CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

        for (int k = 0; k < max_iterations; k++)
        {
            iterations++;
        }

        return iterations;
    }

    // R = B - AX as GEMM:
    // R = -1.0 * AX + R where R initially contains B
    void get_R(hipblasHandle_t &cublasH, float *h_R, const int m, const int n, const float *A, const float *X, const float *B)
    {
        constexpr float alpha = -1;
        constexpr float beta = 1;

        float *d_A = nullptr;
        float *d_X = nullptr;
        float *d_R = nullptr;

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * m * m));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_X), sizeof(float) * m * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(float) * m * n));

        CUDA_CHECK(hipMemcpy(d_A, A, sizeof(float) * m * m, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_X, X, sizeof(float) * m * n, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_R, B, sizeof(float) * m * n, hipMemcpyHostToDevice));

        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    m, n, m,
                                    &alpha, d_A, m, d_X, m,
                                    &beta, d_R, m));

        CUDA_CHECK(hipMemcpy(h_R, d_R, sizeof(float) * m * n, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_X));
        CUDA_CHECK(hipFree(d_R));
    }

    void qr_factorization(hipsolverHandle_t &cusolverH, hipsolverDnParams_t &params, float *Q, float *R, const int m, const int n, const float *A)
    {
        int k = std::min(m, n);
        std::vector<float> tau(k, 0);
        int info = 0;

        float *d_A = nullptr;
        float *d_tau = nullptr;
        int *d_info = nullptr;

        size_t lwork_geqrf_d = 0;
        void *d_work = nullptr;
        size_t lwork_geqrf_h = 0;
        void *h_work = nullptr;

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * m * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_tau), sizeof(float) * tau.size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

        CUDA_CHECK(hipMemcpy(d_A, A, sizeof(float) * m * n, hipMemcpyHostToDevice));

        CUSOLVER_CHECK(hipsolverDnXgeqrf_bufferSize(cusolverH, params, m, n, HIP_R_32F, d_A,
                                                   m, HIP_R_32F, d_tau,
                                                   HIP_R_32F, &lwork_geqrf_d,
                                                   &lwork_geqrf_h));

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), lwork_geqrf_d));

        if (0 < lwork_geqrf_h)
        {
            h_work = reinterpret_cast<void *>(malloc(lwork_geqrf_h));
            if (h_work == nullptr)
            {
                throw std::runtime_error("Error: h_work not allocated.");
            }
        }

        CUSOLVER_CHECK(hipsolverDnXgeqrf(cusolverH, params, m, n, HIP_R_32F, d_A,
                                        m, HIP_R_32F, d_tau,
                                        HIP_R_32F, d_work, lwork_geqrf_d, h_work,
                                        lwork_geqrf_h, d_info));
        free(h_work); // No longer needed

        // Copy R to host (stored in upper triangular)
        CUDA_CHECK(hipMemcpy(R, d_A, sizeof(float) * n * n, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipMemcpy(tau.data(), d_tau, sizeof(float) * tau.size(), hipMemcpyDeviceToHost));

        CUDA_CHECK(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
        if (0 > info)
        {
            std::printf("%d-th parameter is wrong \n", -info);
            exit(1);
        }

        // Explicitly compute Q
        int lwork_orgqr = 0;
        CUSOLVER_CHECK(hipsolverDnSorgqr_bufferSize(cusolverH, m, n, k, d_A, m, d_tau, &lwork_orgqr));
        CUSOLVER_CHECK(hipsolverDnSorgqr(cusolverH, m, n, k, d_A, m, d_tau, reinterpret_cast<float *>(d_work), lwork_orgqr, d_info));

        // Copy Q to host
        CUDA_CHECK(hipMemcpy(Q, d_A, sizeof(float) * m * n, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_info));
        CUDA_CHECK(hipFree(d_tau));
        CUDA_CHECK(hipFree(d_work));
    }
}
